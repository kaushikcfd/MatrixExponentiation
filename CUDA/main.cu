
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdio.h>



__global__ void matrixExponentiation(float *A,float *C, float *D, float *E, float *Coeff, unsigned *N, unsigned *rowsPerThread)
{
	unsigned i,j,k,row,order;

	/*Initializing by the identity matrix. And initialized the matrix C by matrix A.*/
	for(i=0;i<(*rowsPerThread);i++)
	{
		row	=	blockIdx.x*blockDim.x	+	threadIdx.x*(*rowsPerThread)	+	i;
		for(j=0;j<(*N);j++)
		{
			E[row*(*N)+j]	=	Coeff[1]*A[row*(*N)+j];
			D[row*(*N)+j]	=	A[row*(*N)+j];
		}
		E[row*(*N)+row]	+=	Coeff[0];
	}
	__syncthreads();

	for(order=2;order<12;order++)
	{

		for(i=0;i<(*rowsPerThread);i++)
		{
			row	=	blockIdx.x*blockDim.x	+	threadIdx.x*(*rowsPerThread)	+	i;
			for(j=0;j<(*N);j++)
			{
				C[row*(*N)+j]	=	0.0;
				for(k=0;k<(*N);k++)
					C[row*(*N)+j]+=(D[row*(*N)+k]*A[k*(*N)+j]);
			}
		}

		__syncthreads();
	
	
		for(i=0;i<(*rowsPerThread);i++)
		{
			row	=	blockIdx.x*blockDim.x	+	threadIdx.x*(*rowsPerThread)	+	i;
			for(j=0;j<(*N);j++)
			{
				E[row*(*N)+j]+=(Coeff[order]*C[row*(*N)+j]);
				D[row*(*N)+j] =(C[row*(*N)+j]);	
			}
		}
		__syncthreads();
	}
	
}

void makeA(float *A,unsigned N)
{
	unsigned i,j;
	for(i=0;i<N;i++)
		for(j=0;j<N;j++)
			A[i*N+j]=1e-3;	
	return ;
}

void printMatrix(float *A, unsigned N)
{
	unsigned i,j;
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
			printf("%6.4f\t",A[i*N+j]);
		printf("\n");
	}
	return ;
}

int main()
{
	unsigned N,blocks, threads,i;
	unsigned *dev_N;
	float *A,*Exp,*Coeff;
	float *dev_A,*dev_B,*dev_C,*dev_Exp,*dev_Coeff;
	unsigned size;
	unsigned rowsPerThread, *dev_rowsPerThread;	

	printf("The order of matrix to be used\n");
	scanf("%d",&N);	
	printf("Enter the number of blocks.\n");
	scanf("%d",&blocks);
	printf("Enter the number of threads per block.\n");
	scanf("%d",&threads);
	
	if((N%(threads*blocks))!=0)
	{
		printf("The order of the matrix `N` must be divisible by the product `threads*blocks`\n. Aborting the program!\n");
	}
	else
	{
		hipEvent_t	start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		size		=	N*N*sizeof(float);	
		rowsPerThread	=	(N/(threads*blocks));
				
		A		=	(float *)malloc(size);
		Exp		=	(float *)malloc(size);
		Coeff		=	(float *)malloc(12*sizeof(float));
		Coeff[0]	=	1.0;
		for(i=1;i<12;i++)
			Coeff[i]	=	(Coeff[i-1]/(1.0*i));
		hipEventRecord(start);
		hipMalloc((void**)&dev_A,size);
		hipMalloc((void**)&dev_B,size);
		hipMalloc((void**)&dev_C,size);
		hipMalloc((void**)&dev_Exp,size);
		hipMalloc((void**)&dev_Coeff,12*sizeof(float));
		hipMalloc((void**)&dev_rowsPerThread,sizeof(unsigned));
		hipMalloc((void**)&dev_N,sizeof(unsigned));

		makeA(A,N);

		hipMemcpy(dev_A,	A,	size,hipMemcpyHostToDevice);	
		hipMemcpy(dev_Coeff,	Coeff,	12*sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(dev_rowsPerThread,&rowsPerThread,sizeof(unsigned),hipMemcpyHostToDevice);
		hipMemcpy(dev_N,&N,sizeof(unsigned),hipMemcpyHostToDevice);		

		matrixExponentiation<<<blocks,threads>>>(dev_A, dev_B, dev_C, dev_Exp, dev_Coeff ,dev_N,dev_rowsPerThread);		

		hipMemcpy(Exp,dev_Exp,size,hipMemcpyDeviceToHost);

		hipFree(dev_A);
		hipFree(dev_B);
		hipFree(dev_C);
		hipFree(dev_rowsPerThread);
		hipFree(dev_N);

		hipEventRecord(stop);
		hipEventSynchronize(stop);
		float	milliseconds	=	0.0;
		hipEventElapsedTime(&milliseconds,start,stop);
		fprintf (stderr,"Time for the Matrix Multiplication of order %d : %f s using blocks %d and threads per block %d.\n\n",N ,0.001*milliseconds,blocks,threads);
		/*freopen("A.dat","w",stdout);
		printMatrix(A,N);
		fclose(stdout);
		freopen("B.dat","w",stdout);
		printMatrix(B,N);
		fclose(stdout);*/
		freopen("C.dat","w",stdout);
		printMatrix(Exp,N);
		fclose(stdout);
	}
	return 0;
}

